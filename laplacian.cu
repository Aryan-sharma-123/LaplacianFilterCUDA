#include "hip/hip_runtime.h"
%%writefile laplacian.cu
#include <stdio.h>
#include <opencv2/core/core.hpp> // Include core functionalities
#include <opencv2/imgcodecs.hpp> // For imread and imwrite
#include <opencv2/highgui/highgui.hpp> // For GUI functionalities, might not be necessary for this script

using namespace cv; // Use the cv namespace to simplify code

__global__ void boxFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   // only threads inside image will write results
   if((x>=3/2) && (x<(width-3/2)) && (y>=3/2) && (y<(height-3/2)))
   {
      for(int c=0 ; c<channel ; c++)
      {
         // Sum of pixel values
         float sum = 0;
         // Number of filter pixels
         float kS = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-3/2; ky<=3/2; ky++) {
            for(int kx=-3/2; kx<=3/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))*channel+c];
               sum += fl;
               kS += 1;
            }
         }
         dstImage[(y*width+x)*channel+c] =  sum / kS;
      }
   }
}

__global__ void sharpeningFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   float kernel[3][3] = {-1, -1, -1, -1, 9, -1, -1, -1, -1};
   // only threads inside image will write results
   if((x>=3/2) && (x<(width-3/2)) && (y>=3/2) && (y<(height-3/2)))
   {
      for(int c=0 ; c<channel ; c++)
      {
         // Sum of pixel values
         float sum = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-3/2; ky<=3/2; ky++) {
            for(int kx=-3/2; kx<=3/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))*channel+c];
               sum += fl*kernel[ky+3/2][kx+3/2];
            }
         }
         dstImage[(y*width+x)*channel+c] =  sum;
      }
   }
}

void checkCudaErrors(hipError_t r) {
    if (r != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }
}

int main() {
    Mat image = imread("images.jpg", IMREAD_GRAYSCALE);
    if (image.empty()) {
        printf("Error: Image not found.\n");
        return -1;
    }
    int width = image.cols;
    int height = image.rows;
    int channel=image.step/image.cols;
    size_t imageSize = width * height * sizeof(unsigned char);

    unsigned char *h_outputImage = (unsigned char *)malloc(imageSize);
    if (h_outputImage == nullptr) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return -1;
    }

    unsigned char *d_inputImage, *d_outputImage;
    checkCudaErrors(hipMalloc(&d_inputImage, imageSize));
    checkCudaErrors(hipMalloc(&d_outputImage, imageSize));
    checkCudaErrors(hipMemcpy(d_inputImage, image.data, imageSize, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 gridSize(ceil(width/16.0),ceil(height/16.0));
    boxFilter<<<gridSize,blockSize>>>(d_inputImage,d_outputImage,width,height,channel);
    sharpeningFilter<<<gridSize,blockSize>>>(d_outputImage,d_inputImage,width,height,channel);
    checkCudaErrors(hipMemcpy(h_outputImage, d_inputImage, imageSize, hipMemcpyDeviceToHost));

    Mat outputImage(height, width, CV_8UC1, h_outputImage);
    imwrite("output.jpeg", outputImage);

    free(h_outputImage);
    hipFree(d_inputImage);
    hipFree(d_outputImage);

    return 0;
}